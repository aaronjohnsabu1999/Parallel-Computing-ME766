
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// Matrix Multiplication
__global__ void matrixMul(float *A, float *B, float *C, int N)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int k;
  
  if (i < N && j < N)
  {
    float temp = 0;
    for (k = 0; k < N; k++)
      temp += A[i*N + k] * B[k*N + j];
    C[i*N + k] = temp;
  }
}

int main(int argc, char *argv[])
{
  int N = strtol(argv[1], (char **)NULL, 10);
  size_t size = N*N*sizeof(float);
  float *A, *B, *C;
  
  hipMallocManaged(&A, size);
  hipMallocManaged(&B, size);
  hipMallocManaged(&C, size);
  
  // Random Initialization
  for (int i = 0; i < N*N; i++)
  {
    A[i] = rand() / 1000000.0;
    B[i] = rand() / 1000000.0;
  }
  
  // Grid Dimensions
  int threads = 10 * 10;
  int blocks  = (N + threads - 1) / threads;
  // Kernel Launch Parameters
  dim3 THREADS (threads, threads);
  dim3 BLOCKS  ( blocks,  blocks);
  // Launch Kernel
  matrixMul<<<BLOCKS, THREADS>>>(A, B, C, N);
  hipDeviceSynchronize();
  
  // Print Command
  printf("Time taken for CUDA implementation with (N = \t%d) = ", N);
  
  return 0;
}

  